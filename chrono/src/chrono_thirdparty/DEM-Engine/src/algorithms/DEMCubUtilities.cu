//  Copyright (c) 2021, SBEL GPU Development Team
//  Copyright (c) 2021, University of Wisconsin - Madison
//
//	SPDX-License-Identifier: BSD-3-Clause

#include <hipcub/hipcub.hpp>
#include <core/utils/JitHelper.h>
#include <algorithms/DEMCubBasedSubroutines.h>

#include <core/utils/GpuError.h>
#include <algorithms/DEMCubWrappers.cu>

namespace deme {

// These functions interconnecting the cub-part and cpp-part of the code cannot be templated... because of cmake
// restrictions. Not much that I can do, other than writing them all out.

void doubleSumReduce(double* d_in, double* d_out, size_t n, hipStream_t& this_stream, DEMSolverStateData& scratchPad) {
    cubDEMSum<double, double, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}
void floatSumReduce(float* d_in, float* d_out, size_t n, hipStream_t& this_stream, DEMSolverStateData& scratchPad) {
    cubDEMSum<float, float, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

void boolSumReduce(notStupidBool_t* d_in,
                   size_t* d_out,
                   size_t n,
                   hipStream_t& this_stream,
                   DEMSolverStateData& scratchPad) {
    cubDEMSum<notStupidBool_t, size_t, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

void boolMaxReduce(notStupidBool_t* d_in,
                   notStupidBool_t* d_out,
                   size_t n,
                   hipStream_t& this_stream,
                   DEMSolverStateData& scratchPad) {
    cubDEMMax<notStupidBool_t, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

void floatMaxReduce(float* d_in, float* d_out, size_t n, hipStream_t& this_stream, DEMSolverStateData& scratchPad) {
    cubDEMMax<float, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}
void doubleMaxReduce(double* d_in, double* d_out, size_t n, hipStream_t& this_stream, DEMSolverStateData& scratchPad) {
    cubDEMMax<double, DEMSolverStateData>(d_in, d_out, n, this_stream, scratchPad);
}

void floatSumReduceByKey(notStupidBool_t* d_keys_in,
                         notStupidBool_t* d_unique_out,
                         float* d_vals_in,
                         float* d_aggregates_out,
                         size_t* d_num_out,
                         size_t n,
                         hipStream_t& this_stream,
                         DEMSolverStateData& scratchPad) {
    // I'm not sure how to pass cuda hipcub::Sum() as a template argument here, so I used a custom add...
    CubFloatAdd add_op;
    cubDEMReduceByKeys<notStupidBool_t, float, CubFloatAdd, DEMSolverStateData>(
        d_keys_in, d_unique_out, d_vals_in, d_aggregates_out, d_num_out, add_op, n, this_stream, scratchPad);
}
void doubleSumReduceByKey(notStupidBool_t* d_keys_in,
                          notStupidBool_t* d_unique_out,
                          double* d_vals_in,
                          double* d_aggregates_out,
                          size_t* d_num_out,
                          size_t n,
                          hipStream_t& this_stream,
                          DEMSolverStateData& scratchPad) {
    // I'm not sure how to pass cuda hipcub::Sum() as a template argument here, so I used a custom add...
    CubFloatAdd add_op;
    cubDEMReduceByKeys<notStupidBool_t, double, CubFloatAdd, DEMSolverStateData>(
        d_keys_in, d_unique_out, d_vals_in, d_aggregates_out, d_num_out, add_op, n, this_stream, scratchPad);
}

void floatSortByKey(notStupidBool_t* d_keys_in,
                    notStupidBool_t* d_keys_out,
                    float* d_vals_in,
                    float* d_vals_out,
                    size_t n,
                    hipStream_t& this_stream,
                    DEMSolverStateData& scratchPad) {
    cubDEMSortByKeys<notStupidBool_t, float, DEMSolverStateData>(d_keys_in, d_keys_out, d_vals_in, d_vals_out, n,
                                                                 this_stream, scratchPad);
}
void doubleSortByKey(notStupidBool_t* d_keys_in,
                     notStupidBool_t* d_keys_out,
                     double* d_vals_in,
                     double* d_vals_out,
                     size_t n,
                     hipStream_t& this_stream,
                     DEMSolverStateData& scratchPad) {
    cubDEMSortByKeys<notStupidBool_t, double, DEMSolverStateData>(d_keys_in, d_keys_out, d_vals_in, d_vals_out, n,
                                                                  this_stream, scratchPad);
}

}  // namespace deme
