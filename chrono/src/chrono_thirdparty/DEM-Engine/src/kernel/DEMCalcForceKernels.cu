#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
#include <DEM/Defines.h>
#include <kernel/DEMHelperKernels.cu>
#include <kernel/DEMCollisionKernels.cu>

// If clump templates are jitified, they will be below
_clumpTemplateDefs_;
// Definitions of analytical entites are below
_analyticalEntityDefs_;
// Material properties are below
_materialDefs_;
// If mass properties are jitified, then they are below
_massDefs_;

template <typename T1>
inline __device__ void equipOwnerPosRot(deme::DEMDataDT* granData,
                                        const deme::bodyID_t& myOwner,
                                        T1& relPos,
                                        double3& ownerPos,
                                        double3& bodyPos,
                                        float4& oriQ) {
    voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
        ownerPos.x, ownerPos.y, ownerPos.z, granData->voxelID[myOwner], granData->locX[myOwner],
        granData->locY[myOwner], granData->locZ[myOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
    oriQ.w = granData->oriQw[myOwner];
    oriQ.x = granData->oriQx[myOwner];
    oriQ.y = granData->oriQy[myOwner];
    oriQ.z = granData->oriQz[myOwner];
    applyOriQToVector3(relPos.x, relPos.y, relPos.z, oriQ.w, oriQ.x, oriQ.y, oriQ.z);
    bodyPos.x = ownerPos.x + (double)relPos.x;
    bodyPos.y = ownerPos.y + (double)relPos.y;
    bodyPos.z = ownerPos.z + (double)relPos.z;
}

__global__ void calculateContactForces(deme::DEMSimParams* simParams, deme::DEMDataDT* granData, size_t nContactPairs) {
    deme::contactPairs_t myContactID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myContactID < nContactPairs) {
        // Identify contact type first
        deme::contact_t myContactType = granData->contactType[myContactID];
        // The following quantities are always calculated, regardless of force model
        double3 contactPnt;
        float3 B2A;  // Unit vector pointing from body B to body A (contact normal)
        double overlapDepth;
        double3 AOwnerPos, bodyAPos, BOwnerPos, bodyBPos;
        float AOwnerMass, ARadius, BOwnerMass, BRadius;
        float4 AOriQ, BOriQ;
        deme::materialsOffset_t bodyAMatType, bodyBMatType;
        // Then allocate the optional quantities that will be needed in the force model (note: this one can't be in a
        // curly bracket, obviously...)
        _forceModelIngredientDefinition_;
        // Take care of 2 bodies in order, bodyA first, grab location and velocity to local cache
        // We know in this kernel, bodyA will be a sphere; bodyB can be something else
        {
            deme::bodyID_t sphereID = granData->idGeometryA[myContactID];
            deme::bodyID_t myOwner = granData->ownerClumpBody[sphereID];

            float3 myRelPos;
            float myRadius;
            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPos, myRadius
            // Use an input named exactly `sphereID' which is the id of this sphere component
            { _componentAcqStrat_; }

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                _massAcqStrat_;
                AOwnerMass = myMass;
            }

            equipOwnerPosRot(granData, myOwner, myRelPos, AOwnerPos, bodyAPos, AOriQ);

            ARadius = myRadius;
            bodyAMatType = granData->sphereMaterialOffset[sphereID];

            // Optional force model ingredients are loaded here...
            _forceModelIngredientAcqForA_;
        }

        // Then bodyB, location and velocity
        if (myContactType == deme::SPHERE_SPHERE_CONTACT) {
            deme::bodyID_t sphereID = granData->idGeometryB[myContactID];
            deme::bodyID_t myOwner = granData->ownerClumpBody[sphereID];

            float3 myRelPos;
            float myRadius;
            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPos, myRadius
            // Use an input named exactly `sphereID' which is the id of this sphere component
            { _componentAcqStrat_; }

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                _massAcqStrat_;
                BOwnerMass = myMass;
            }

            equipOwnerPosRot(granData, myOwner, myRelPos, BOwnerPos, bodyBPos, BOriQ);

            BRadius = myRadius;
            bodyBMatType = granData->sphereMaterialOffset[sphereID];

            _forceModelIngredientAcqForB_;

            myContactType = checkSpheresOverlap<double, float>(
                bodyAPos.x, bodyAPos.y, bodyAPos.z, ARadius, bodyBPos.x, bodyBPos.y, bodyBPos.z, BRadius, contactPnt.x,
                contactPnt.y, contactPnt.z, B2A.x, B2A.y, B2A.z, overlapDepth);
        } else if (myContactType == deme::SPHERE_MESH_CONTACT) {
            deme::bodyID_t triB = granData->idGeometryB[myContactID];
            deme::bodyID_t myOwner = granData->ownerMesh[triB];
            //// TODO: Is this OK?
            BRadius = DEME_HUGE_FLOAT;
            bodyBMatType = granData->triMaterialOffset[triB];

            double3 triNode1 = to_double3(granData->relPosNode1[triB]);
            double3 triNode2 = to_double3(granData->relPosNode2[triB]);
            double3 triNode3 = to_double3(granData->relPosNode3[triB]);

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            {
                float myMass;
                _massAcqStrat_;
                BOwnerMass = myMass;
            }

            // bodyBPos is for a place holder for the outcome triNode1 position
            equipOwnerPosRot(granData, myOwner, triNode1, BOwnerPos, bodyBPos, BOriQ);
            triNode1 = bodyBPos;
            // Do this to node 2 and 3 as well
            applyOriQToVector3(triNode2.x, triNode2.y, triNode2.z, BOriQ.w, BOriQ.x, BOriQ.y, BOriQ.z);
            triNode2 += BOwnerPos;
            applyOriQToVector3(triNode3.x, triNode3.y, triNode3.z, BOriQ.w, BOriQ.x, BOriQ.y, BOriQ.z);
            triNode3 += BOwnerPos;
            // Assign the correct bodyBPos
            bodyBPos = triangleCentroid<double3>(triNode1, triNode2, triNode3);

            _forceModelIngredientAcqForB_;

            double3 contact_normal;
            bool in_contact = triangle_sphere_CD<double3, double>(triNode1, triNode2, triNode3, bodyAPos, ARadius,
                                                                  contact_normal, overlapDepth, contactPnt);
            B2A = to_float3(contact_normal);
            overlapDepth = -overlapDepth;  // triangle_sphere_CD gives neg. number for overlapping cases

            // If not in contact, correct myContactType
            if (!in_contact) {
                myContactType = deme::NOT_A_CONTACT;
            }
        } else {
            // If B is analytical entity, its owner, relative location, material info is jitified
            deme::objID_t bodyB = granData->idGeometryB[myContactID];
            deme::bodyID_t myOwner = objOwner[bodyB];
            bodyBMatType = objMaterial[bodyB];
            BOwnerMass = objMass[bodyB];
            //// TODO: Is this OK?
            BRadius = DEME_HUGE_FLOAT;
            float3 myRelPos;
            float3 bodyBRot;
            myRelPos.x = objRelPosX[bodyB];
            myRelPos.y = objRelPosY[bodyB];
            myRelPos.z = objRelPosZ[bodyB];

            equipOwnerPosRot(granData, myOwner, myRelPos, BOwnerPos, bodyBPos, BOriQ);

            // B's orientation (such as plane normal) is rotated with its owner too
            bodyBRot.x = objRotX[bodyB];
            bodyBRot.y = objRotY[bodyB];
            bodyBRot.z = objRotZ[bodyB];
            applyOriQToVector3<float, deme::oriQ_t>(bodyBRot.x, bodyBRot.y, bodyBRot.z, BOriQ.w, BOriQ.x, BOriQ.y,
                                                    BOriQ.z);

            _forceModelIngredientAcqForB_;

            // Note for this test on dT side we don't enlarge entities
            myContactType = checkSphereEntityOverlap<double3, float, double>(
                bodyAPos, ARadius, objType[bodyB], bodyBPos, bodyBRot, objSize1[bodyB], objSize2[bodyB],
                objSize3[bodyB], objNormal[bodyB], 0.0, contactPnt, B2A, overlapDepth);
        }

        _forceModelContactWildcardAcq_;
        if (myContactType != deme::NOT_A_CONTACT) {
            float3 force = make_float3(0, 0, 0);
            float3 torque_only_force = make_float3(0, 0, 0);
            // Local position of the contact point is always a piece of info we require... regardless of force model
            float3 locCPA = to_float3(contactPnt - AOwnerPos);
            float3 locCPB = to_float3(contactPnt - BOwnerPos);
            // Now map this contact point location to bodies' local ref
            applyOriQToVector3<float, deme::oriQ_t>(locCPA.x, locCPA.y, locCPA.z, AOriQ.w, -AOriQ.x, -AOriQ.y,
                                                    -AOriQ.z);
            applyOriQToVector3<float, deme::oriQ_t>(locCPB.x, locCPB.y, locCPB.z, BOriQ.w, -BOriQ.x, -BOriQ.y,
                                                    -BOriQ.z);
            // The following part, the force model, is user-specifiable
            // NOTE!! "force" and "delta_tan" and "delta_time" must be properly set by this piece of code
            { _DEMForceModel_; }

            // Write contact location values back to global memory
            _contactInfoWrite_;

            // If force model modifies owner wildcards, write them back here
            _forceModelOwnerWildcardWrite_;

            // Optionally, the forces can be reduced to acc right here (may be faster for polydisperse spheres)
            _forceCollectInPlaceStrat_;
        } else {
            // The contact is no longer active, so we need to destroy its contact history recording
            _forceModelContactWildcardDestroy_;
        }

        // Updated contact wildcards need to be write back to global mem. It is here because contact wildcard may need
        // to be destroyed for non-contact, so it has to go last.
        _forceModelContactWildcardWrite_;
    }
}
