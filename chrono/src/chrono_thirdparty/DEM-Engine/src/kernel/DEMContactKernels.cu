#include "hip/hip_runtime.h"
// DEM contact detection-related custom kernels
#include <DEM/Defines.h>
#include <kernel/DEMHelperKernels.cu>

#include <cub/util_ptx.cuh>

// If clump templates are jitified, they will be below
_clumpTemplateDefs_;
// Family mask, _nFamilyMaskEntries_ elements are in this array
// __constant__ __device__ bool familyMasks[] = {_familyMasks_};

__global__ void getNumberOfSphereContactsEachBin(deme::DEMSimParams* simParams,
                                                 deme::DEMDataKT* granData,
                                                 deme::bodyID_t* sphereIDsEachBinTouches_sorted,
                                                 deme::binID_t* activeBinIDs,
                                                 deme::spheresBinTouches_t* numSpheresBinTouches,
                                                 deme::binSphereTouchPairs_t* sphereIDsLookUpTable,
                                                 deme::spheresBinTouches_t* numContactsInEachBin,
                                                 size_t nActiveBins) {
    // Only active bins got execute this...
    deme::binID_t myActiveID = blockIdx.x * blockDim.x + threadIdx.x;
    // I need to store all the sphereIDs that I am supposed to look into
    // A100 has about 164K shMem... these arrays really need to be small, or we can only fit a small number of bins in
    // one block
    deme::bodyID_t ownerIDs[DEME_MAX_SPHERES_PER_BIN];
    float radii[DEME_MAX_SPHERES_PER_BIN];
    double bodyX[DEME_MAX_SPHERES_PER_BIN];
    double bodyY[DEME_MAX_SPHERES_PER_BIN];
    double bodyZ[DEME_MAX_SPHERES_PER_BIN];
    deme::family_t ownerFamily[DEME_MAX_SPHERES_PER_BIN];
    if (myActiveID < nActiveBins) {
        // I got a true bin ID
        deme::binID_t binID = activeBinIDs[myActiveID];

        deme::spheresBinTouches_t contact_count = 0;
        // Grab the bodies that I care, put into local memory
        deme::spheresBinTouches_t nBodiesMeHandle = numSpheresBinTouches[myActiveID];
        if (nBodiesMeHandle > DEME_MAX_SPHERES_PER_BIN) {
            DEME_ABORT_KERNEL("Bin %u contains %u sphere components, exceeding maximum allowance (%u)\n", myActiveID,
                              nBodiesMeHandle, DEME_MAX_SPHERES_PER_BIN);
        }

        deme::binSphereTouchPairs_t myBodiesTableEntry = sphereIDsLookUpTable[myActiveID];
        // printf("nBodies: %u\n", nBodiesMeHandle);
        for (deme::spheresBinTouches_t i = 0; i < nBodiesMeHandle; i++) {
            deme::bodyID_t sphereID = sphereIDsEachBinTouches_sorted[myBodiesTableEntry + i];
            ownerIDs[i] = granData->ownerClumpBody[sphereID];
            ownerFamily[i] = granData->familyID[ownerIDs[i]];
            double ownerX, ownerY, ownerZ;
            float myRadius;
            float3 myRelPos;

            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPos, myRadius (in CD kernels, radius needs to be expanded)
            // Use an input named exactly `sphereID' which is the id of this sphere component
            {
                _componentAcqStrat_;
                myRadius += simParams->beta;
            }

            voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
                ownerX, ownerY, ownerZ, granData->voxelID[ownerIDs[i]], granData->locX[ownerIDs[i]],
                granData->locY[ownerIDs[i]], granData->locZ[ownerIDs[i]], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            float myOriQw = granData->oriQw[ownerIDs[i]];
            float myOriQx = granData->oriQx[ownerIDs[i]];
            float myOriQy = granData->oriQy[ownerIDs[i]];
            float myOriQz = granData->oriQz[ownerIDs[i]];
            applyOriQToVector3<float, deme::oriQ_t>(myRelPos.x, myRelPos.y, myRelPos.z, myOriQw, myOriQx, myOriQy,
                                                    myOriQz);
            bodyX[i] = ownerX + (double)myRelPos.x;
            bodyY[i] = ownerY + (double)myRelPos.y;
            bodyZ[i] = ownerZ + (double)myRelPos.z;
            radii[i] = myRadius;
        }

        for (deme::spheresBinTouches_t bodyA = 0; bodyA < nBodiesMeHandle; bodyA++) {
            for (deme::spheresBinTouches_t bodyB = bodyA + 1; bodyB < nBodiesMeHandle; bodyB++) {
                // For 2 bodies to be considered in contact, the contact point must be in this bin (to avoid
                // double-counting), and they do not belong to the same clump
                if (ownerIDs[bodyA] == ownerIDs[bodyB])
                    continue;

                // Grab family number from memory (not jitified: b/c family number can change frequently in a sim)
                unsigned int bodyAFamily = ownerFamily[bodyA];
                unsigned int bodyBFamily = ownerFamily[bodyB];
                unsigned int maskMatID = locateMaskPair<unsigned int>(bodyAFamily, bodyBFamily);
                // If marked no contact, skip ths iteration
                if (granData->familyMasks[maskMatID] != deme::DONT_PREVENT_CONTACT) {
                    continue;
                }

                double contactPntX;
                double contactPntY;
                double contactPntZ;
                bool in_contact;
                in_contact = checkSpheresOverlap<double>(bodyX[bodyA], bodyY[bodyA], bodyZ[bodyA], radii[bodyA],
                                                         bodyX[bodyB], bodyY[bodyB], bodyZ[bodyB], radii[bodyB],
                                                         contactPntX, contactPntY, contactPntZ);
                deme::binID_t contactPntBin = getPointBinID<deme::binID_t>(
                    contactPntX, contactPntY, contactPntZ, simParams->binSize, simParams->nbX, simParams->nbY);

                /*
                printf("contactPntBin: %u, %u, %u\n", (unsigned int)(contactPntX/_binSize_),
                                                        (unsigned int)(contactPntY/_binSize_),
                                                        (unsigned int)(contactPntZ/_binSize_));
                unsigned int ZZ = binID/(_nbX_*_nbY_);
                unsigned int YY = binID%(_nbX_*_nbY_)/_nbX_;
                unsigned int XX = binID%(_nbX_*_nbY_)%_nbX_;
                printf("binID: %u, %u, %u\n", XX,YY,ZZ);
                printf("bodyA: %f, %f, %f\n", bodyX[bodyA], bodyY[bodyA], bodyZ[bodyA]);
                printf("bodyB: %f, %f, %f\n", bodyX[bodyB], bodyY[bodyB], bodyZ[bodyB]);
                printf("contactPnt: %f, %f, %f\n", contactPntX, contactPntY, contactPntZ);
                printf("contactPntBin: %u\n", contactPntBin);
                */

                if (in_contact && (contactPntBin == binID)) {
                    contact_count++;
                }
            }
        }
        numContactsInEachBin[myActiveID] = contact_count;
    }
}

__global__ void populateSphSphContactPairsEachBin(deme::DEMSimParams* simParams,
                                                  deme::DEMDataKT* granData,
                                                  deme::bodyID_t* sphereIDsEachBinTouches_sorted,
                                                  deme::binID_t* activeBinIDs,
                                                  deme::spheresBinTouches_t* numSpheresBinTouches,
                                                  deme::binSphereTouchPairs_t* sphereIDsLookUpTable,
                                                  deme::contactPairs_t* contactReportOffsets,
                                                  deme::bodyID_t* idSphA,
                                                  deme::bodyID_t* idSphB,
                                                  deme::contact_t* dType,
                                                  size_t nActiveBins) {
    // Only active bins got to execute this...
    deme::binID_t myActiveID = blockIdx.x * blockDim.x + threadIdx.x;
    // I need to store all the sphereIDs that I am supposed to look into
    // A100 has about 164K shMem... these arrays really need to be small, or we can only fit a small number of bins in
    // one block
    deme::bodyID_t ownerIDs[DEME_MAX_SPHERES_PER_BIN];
    deme::bodyID_t bodyIDs[DEME_MAX_SPHERES_PER_BIN];
    float radii[DEME_MAX_SPHERES_PER_BIN];
    double bodyX[DEME_MAX_SPHERES_PER_BIN];
    double bodyY[DEME_MAX_SPHERES_PER_BIN];
    double bodyZ[DEME_MAX_SPHERES_PER_BIN];
    deme::family_t ownerFamily[DEME_MAX_SPHERES_PER_BIN];
    if (myActiveID < nActiveBins) {
        // But I got a true bin ID
        deme::binID_t binID = activeBinIDs[myActiveID];

        // Grab the bodies that I care, put into local memory
        deme::spheresBinTouches_t nBodiesMeHandle = numSpheresBinTouches[myActiveID];
        deme::binSphereTouchPairs_t myBodiesTableEntry = sphereIDsLookUpTable[myActiveID];
        for (deme::spheresBinTouches_t i = 0; i < nBodiesMeHandle; i++) {
            deme::bodyID_t sphereID = sphereIDsEachBinTouches_sorted[myBodiesTableEntry + i];
            ownerIDs[i] = granData->ownerClumpBody[sphereID];
            ownerFamily[i] = granData->familyID[ownerIDs[i]];
            bodyIDs[i] = sphereID;
            double ownerX, ownerY, ownerZ;
            float myRadius;
            float3 myRelPos;

            // Get my component offset info from either jitified arrays or global memory
            // Outputs myRelPos, myRadius (in CD kernels, radius needs to be expanded)
            // Use an input named exactly `sphereID' which is the id of this sphere component
            {
                _componentAcqStrat_;
                myRadius += simParams->beta;
            }

            voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
                ownerX, ownerY, ownerZ, granData->voxelID[ownerIDs[i]], granData->locX[ownerIDs[i]],
                granData->locY[ownerIDs[i]], granData->locZ[ownerIDs[i]], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            float myOriQw = granData->oriQw[ownerIDs[i]];
            float myOriQx = granData->oriQx[ownerIDs[i]];
            float myOriQy = granData->oriQy[ownerIDs[i]];
            float myOriQz = granData->oriQz[ownerIDs[i]];
            applyOriQToVector3<float, deme::oriQ_t>(myRelPos.x, myRelPos.y, myRelPos.z, myOriQw, myOriQx, myOriQy,
                                                    myOriQz);
            bodyX[i] = ownerX + (double)myRelPos.x;
            bodyY[i] = ownerY + (double)myRelPos.y;
            bodyZ[i] = ownerZ + (double)myRelPos.z;
            radii[i] = myRadius;
        }

        // Get my offset for writing back to the global arrays that contain contact pair info
        deme::contactPairs_t myReportOffset = contactReportOffsets[myActiveID];

        for (deme::spheresBinTouches_t bodyA = 0; bodyA < nBodiesMeHandle; bodyA++) {
            for (deme::spheresBinTouches_t bodyB = bodyA + 1; bodyB < nBodiesMeHandle; bodyB++) {
                // For 2 bodies to be considered in contact, the contact point must be in this bin (to avoid
                // double-counting), and they do not belong to the same clump
                if (ownerIDs[bodyA] == ownerIDs[bodyB])
                    continue;

                // Grab family number from memory (not jitified: b/c family number can change frequently in a sim)
                unsigned int bodyAFamily = ownerFamily[bodyA];
                unsigned int bodyBFamily = ownerFamily[bodyB];
                unsigned int maskMatID = locateMaskPair<unsigned int>(bodyAFamily, bodyBFamily);
                // If marked no contact, skip ths iteration
                if (granData->familyMasks[maskMatID] != deme::DONT_PREVENT_CONTACT) {
                    continue;
                }

                double contactPntX;
                double contactPntY;
                double contactPntZ;
                bool in_contact;
                in_contact = checkSpheresOverlap<double>(bodyX[bodyA], bodyY[bodyA], bodyZ[bodyA], radii[bodyA],
                                                         bodyX[bodyB], bodyY[bodyB], bodyZ[bodyB], radii[bodyB],
                                                         contactPntX, contactPntY, contactPntZ);
                deme::binID_t contactPntBin = getPointBinID<deme::binID_t>(
                    contactPntX, contactPntY, contactPntZ, simParams->binSize, simParams->nbX, simParams->nbY);

                if (in_contact && (contactPntBin == binID)) {
                    idSphA[myReportOffset] = bodyIDs[bodyA];
                    idSphB[myReportOffset] = bodyIDs[bodyB];
                    dType[myReportOffset] = deme::SPHERE_SPHERE_CONTACT;
                    myReportOffset++;
                }
            }
        }
    }
}
